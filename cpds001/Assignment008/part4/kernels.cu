#include <math.h>
#include <float.h>
#include <hip/hip_runtime.h>

__global__ void gpu_Heat (float *h, float *g, float *residual,int N) {
	int i = (blockIdx.x * blockDim.x) + threadIdx.x + 1;
	int j = (blockIdx.y * blockDim.y) + threadIdx.y + 1;
 	if( i < N-1 && j < N-1) {
		int pos = (i*N)+j;
		g[pos]= 0.25 * (h[pos-1] + h[pos+1] + h[pos+N] + h[ pos-N ]);
  		residual[pos] = g[pos] - h[pos];
  		residual[pos] *= residual[pos];
	}
}

__device__ void warpReduce(volatile float* sdata, int tid) {
    sdata[tid] += sdata[tid + 32];
    sdata[tid] += sdata[tid + 16];
    sdata[tid] += sdata[tid + 8];
    sdata[tid] += sdata[tid + 4];
    sdata[tid] += sdata[tid + 2];
    sdata[tid] += sdata[tid + 1];
}

// Slide 22 : http://developer.download.nvidia.com/assets/cuda/files/reduction.pdf 
__global__ void gpu_reduction (float *g_idata, float *g_odata) {
	extern __shared__ float sdata[];
	
	unsigned int tid = threadIdx.x;
	unsigned int i = blockIdx.x*blockDim.x + threadIdx.x;
	sdata[tid] = g_idata[i];
	__syncthreads();

	for(unsigned int s=blockDim.x/2; s>32; s>>=1) {
		if (tid < s)
			sdata[tid] += sdata[tid + s];
		__syncthreads();
	}

	if (tid < 32) warpReduce(sdata, tid);
	if (tid == 0) g_odata[blockIdx.x] = sdata[0];
}
